#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include "cuerr.hpp"
#include "cudaconstraits.hpp"
#include "dsimemutil.h"

texture<float, 2> texLeftImg;
texture<float, 2> texRightImg;

__device__ float ccAtDisp(int x, int y, int disp)
{
  float domSum = 0.0f,
    lSum = 0.0f,
    rSum = 0.0f;

  for (int row=-1; row<2; row++)
    for (int col=-1; col<2; col++) {
      const float lValue = tex2D(texLeftImg, x + col, y + row);
      const float rValue = tex2D(texRightImg, x + col - disp, y + row);
      domSum += lValue*rValue;
       
      lSum += lValue*lValue;
      rSum += rValue*rValue;      
    }

  return abs(1.0f - domSum/sqrt(lSum*rSum));
}

__global__ void ccorrelationKern(const dim3 dsiDim, hipPitchedPtr dsiMem)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if ( x < dsiDim.x && y < dsiDim.y ) {
    float *dsiRow = dsiGetRow(dsiMem, dsiDim.x, x, y);
    
    for (int disp=0; disp < dsiDim.z; disp++) {
      float ccValue = HIP_INF_F;
      if ( x - disp >= 0)
        ccValue = ccAtDisp(x, y, disp);
      dsiRow[disp] = ccValue;
    }
    
  }
}

TDV_NAMESPACE_BEGIN

void CrossCorrelationDevRun(Dim dsiDim, float *leftImg_d, float *rightImg_d,
                            hipPitchedPtr dsiMem)
{
  CUerrExp err;
  
  err << hipBindTexture2D(NULL, texLeftImg, leftImg_d,
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));

  err << hipBindTexture2D(NULL, texRightImg, rightImg_d,
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));

  texLeftImg.addressMode[0] = texRightImg.addressMode[0] = hipAddressModeWrap;
  texLeftImg.addressMode[1] = texRightImg.addressMode[1] = hipAddressModeWrap;
  texLeftImg.normalized = texRightImg.normalized = false;
  texLeftImg.filterMode = texRightImg.filterMode = hipFilterModePoint;    
  
  CudaConstraits constraits;
  WorkSize ws = constraits.imageWorkSize(dsiDim);
  ccorrelationKern<<<ws.blocks, ws.threads>>>(tdvDimTo(dsiDim), dsiMem);   
}

TDV_NAMESPACE_END