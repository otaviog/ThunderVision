#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include "dim.hpp"
#include "dsimemutil.h"
#include "benchmark.hpp"
#include "cudaconstraits.hpp"

#include <iostream>

#define MAX_DISP 768

#define min4(a, b, c, d) min(a, min(b, min(c, d)))

__global__ void wtaKernel(const DSIDim dim,
                          const float *dsi,
                          const uint maxOffset,
                          float *outimg);

__global__ void zeroDSIVolume(const DSIDim dsiDim,
                              float *dsiVol)
{
  const uint x = blockDim.x*blockIdx.x + threadIdx.x;
  const uint y = blockDim.y*blockIdx.y + threadIdx.y;

  if ( x < dsiDim.x && y < dsiDim.y ) {
    for (uint z=0; z<dsiDim.z; z++) {
      dsiVol[dsiOffset(dsiDim, x, y, z)] = 0.0f;
    }
  }
}

const float P1 = 30.0f/255.0f;
const float P2 = 150.0f/255.0f;

inline __device__ float pathCost(const DSIDim dsiDim, const float *costDSI,
                                 float lcD, float lcDm1, float lcDp1,
                                 uint x, uint y, uint z,
                                 float minDisp,
                                 float P2Adjust,
                                 float *aggregVol)
{
  const uint dsiOff = dsiOffset(dsiDim, x, y, z);
  const float cost = costDSI[dsiOff];

  const float Lr = cost
    + min4(lcD,
           lcDm1 + P1,
           lcDp1 + P1,
           minDisp + P2Adjust) - minDisp;

  aggregVol[dsiOff] += Lr;

  return Lr;
}

__constant__ int matrix[6];
__constant__ int invMatrix[6];

__constant__ int rowsWidths[512];

__global__ void semiglobalAggregVolKernel(const DSIDim dsiDim,
                                          const uint maxRowWidth,
                                          const float *costDSI,
                                          const float *iImage,
                                          float *aggregDSI)
{
  const uint z = threadIdx.x;
  const uint y = blockIdx.x;
  const uint dz = z + 1;

  __shared__ float prevDisps[2][MAX_DISP + 2];
  __shared__ float prevDispsMin[2][MAX_DISP];
  __shared__ float fMinDisp, iMinDisp;
  __shared__ float fP2Adjust, iP2Adjust,
    fLastIntesity, iLastIntesity;

  __shared__ int fX, fY, iX, iY;

  if ( z == 0 ) {
    fX = matrix[1]*y + matrix[2];
    fY = matrix[4]*y + matrix[5];

    iX = invMatrix[1]*y + invMatrix[2];
    iY = invMatrix[4]*y + invMatrix[5];

    prevDisps[0][0] = HIP_INF_F;
    prevDisps[0][dsiDim.z + 1] = HIP_INF_F;

    prevDisps[1][0] = HIP_INF_F;
    prevDisps[1][dsiDim.z + 1] = HIP_INF_F;

    fLastIntesity = iImage[fY*dsiDim.x + fX];
    iLastIntesity = iImage[iY*dsiDim.x + iX];
  }

  __syncthreads();

  uint initialOff = dsiOffset(dsiDim, fX, fY, z);
  float initialCost = costDSI[initialOff];

  prevDisps[0][dz] = initialCost;
  prevDispsMin[0][z] = initialCost;
  aggregDSI[initialOff] = initialCost;

  initialOff = dsiOffset(dsiDim, iX, iY, z);
  initialCost = costDSI[initialOff];

  prevDisps[1][dz] = initialCost;
  prevDispsMin[1][z] = initialCost;
  aggregDSI[initialOff] = initialCost;

  __syncthreads();

  float fLr, iLr;

  for (uint x=1; x<maxRowWidth; x++) {    
    int i = dsiDim.z >> 1;
    while ( i != 0 ) {
      if ( z < i ) {
        prevDispsMin[0][z] = min(prevDispsMin[0][z],
                                 prevDispsMin[0][z + i]);
        prevDispsMin[1][z] = min(prevDispsMin[1][z],
                                 prevDispsMin[1][z + i]);
      }
      __syncthreads();
      i = i >> 1;
    }

    if ( z == 0 && x < rowsWidths[y] ) {
      fMinDisp = prevDispsMin[0][0];
      iMinDisp = prevDispsMin[1][0];

      fX = matrix[0]*x + matrix[1]*y + matrix[2];
      fY = matrix[3]*x + matrix[4]*y + matrix[5];

      iX = invMatrix[0]*x + invMatrix[1]*y + invMatrix[2];
      iY = invMatrix[3]*x + invMatrix[4]*y + invMatrix[5];

      const float fI = iImage[fY*dsiDim.x + fX];
      const float iI = iImage[iY*dsiDim.x + iX];

      fP2Adjust = P2/abs(fI - fLastIntesity);
      iP2Adjust = P2/abs(iI - iLastIntesity);

      fLastIntesity = fI;
      iLastIntesity = iI;
    }

    __syncthreads();

    if ( x < rowsWidths[y] ) {
      fLr = pathCost(dsiDim, costDSI,
                     prevDisps[0][dz],
                     prevDisps[0][dz - 1],
                     prevDisps[0][dz + 1],
                     fX, fY, z, fMinDisp, fP2Adjust,
                     aggregDSI);

#if 0
      iLr = pathCost(dsiDim, costDSI,
                     prevDisps[1][dz],
                     prevDisps[1][dz - 1],
                     prevDisps[1][dz + 1],
                     iX, iY, z, iMinDisp, iP2Adjust,
                     aggregDSI);
#endif
    }

    __syncthreads();

    prevDisps[0][dz] = prevDispsMin[0][z] = fLr;
    prevDisps[1][dz] = prevDispsMin[1][z] = iLr;

    __syncthreads();
  }
}

void RunSemiGlobalDev(const tdv::Dim &tdv_dsiDim, const float *dsi,
                      const float *lorigin, float *aggregDSI, float *dispImg)
{
  DSIDim dsiDim(DSIDimCreate(tdv_dsiDim));

  const int leftRightM[2][3] = {
    {1, 0, 0},
    {0, 1, 0},
  };

  const int rightLeftM[2][3] = {
    {-1, 0, dsiDim.x - 1},
    {0, 1, 0},
  };

  const int topBottomM[2][3] = {
    {0, 1, 0},
    {1, 0, 0}
  };

  const int bottomTopM[2][3] = {
    {0, 1, 0},
    {-1, 0, dsiDim.y - 1}
  };

  const int leftRightDiagM[2][3] = {
    { -1, 0, dsiDim.x},
    {1, 1, 0},
  };
  
  const int rightLeftDiagM[2][3] = {
    { -1, 0, dsiDim.x},
    {1, 1, 0},
  };

  tdv::CUerrExp cuerr;

  tdv::CudaConstraits constraits;

  tdv::CudaBenchmarker bm;
  bm.begin();
    
  tdv::WorkSize wsz = constraits.imageWorkSize(tdv_dsiDim);
  int rowsWidths_h[512];
  
  zeroDSIVolume<<<wsz.blocks, wsz.threads>>>(dsiDim, aggregDSI);    
  cuerr << hipDeviceSynchronize();
  
  for (size_t i=0; i<512; i++) {
    rowsWidths_h[i] = dsiDim.x;
  }
  
  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(rowsWidths), rowsWidths_h, sizeof(int)*512);
  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(matrix), leftRightM, sizeof(int)*6);
  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(invMatrix), rightLeftM, sizeof(int)*6);
  
  semiglobalAggregVolKernel<<<dsiDim.y, dsiDim.z>>>(dsiDim, dsiDim.x,
                                                    dsi, lorigin,
                                                    aggregDSI);  
  hipDeviceSynchronize();

  for (size_t i=0; i<512; i++) {
    rowsWidths_h[i] = dsiDim.y;
  }
  
  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(rowsWidths), rowsWidths_h, sizeof(int)*512);
  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(matrix), topBottomM, sizeof(int)*6);
  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(invMatrix), bottomTopM, sizeof(int)*6);
  
  semiglobalAggregVolKernel<<<dsiDim.x, dsiDim.z>>>(dsiDim, dsiDim.y,
                                                    dsi, lorigin,
                                                    aggregDSI);
  cuerr << hipDeviceSynchronize();
  
  for (size_t i=0; i<256; i++) {
    rowsWidths_h[i] = i + 1;
  }
  
  for (size_t i=256; i<512; i++) {
    rowsWidths_h[i] = 512 - i;
  }

  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(rowsWidths), rowsWidths_h, sizeof(int)*512);
  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(matrix), leftRightDiagM, sizeof(int)*6);
  cuerr << hipMemcpyToSymbol(HIP_SYMBOL(invMatrix), rightLeftDiagM, sizeof(int)*6);
  
  semiglobalAggregVolKernel<<<dsiDim.x, dsiDim.z>>>(dsiDim, dsiDim.y,
                                                    dsi, lorigin,
                                                    aggregDSI);
  cuerr << hipDeviceSynchronize();
  
  wtaKernel<<<wsz.blocks, wsz.threads>>>(dsiDim, aggregDSI,
                                         dsiDim.x*dsiDim.y,
                                         dispImg);     
  
  std::cout<<bm.end()<<std::endl;
}
