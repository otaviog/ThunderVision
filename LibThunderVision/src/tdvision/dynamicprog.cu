#include "hip/hip_runtime.h"
#include "dim.hpp"
#include "dsimem.hpp"
#include "dsimemutil.h"
#include "benchmark.hpp"
#include "cudaconstraits.hpp"

#define SINGLE_KERNEL 1

#ifdef SINGLE_KERNEL
__global__ void dynamicprog(const DSIDim dim, const float *costDSI,
                            float *sumCostDSI, int *pathDSI)
{
  //uint z = threadIdx.x + blockIdx.x*blockDim.x;
  //uint y = threadIdx.y + blockIdx.y*blockDim.y;
  const uint z = threadIdx.x;
  const uint y = blockIdx.x;

  if ( z >= dim.z || y >= dim.y  )
    return ;
  
  const uint initialOff = dsiOffset(dim, 0, y, z);
  sumCostDSI[initialOff] = costDSI[initialOff];
  sumCostDSI[initialOff] = 0;
  __threadfence();
  pathDSI[initialOff] = 0;    
  __syncthreads();
  
  __shared__ float sharedCost[32];    
  for (uint x=0; x<dim.x; x++) {    
    
    const uint c0Offset = dsiOffset(dim, x, y, z);  
    const float c0 = costDSI[c0Offset];
    float c2;
    if ( x > 0 )
      c2 = sumCostDSI[dsiOffset(dim, x - 1, y, z)];
    else
      c2 = c0;
    
    sharedCost[z] = c2;
    __syncthreads();
    
    float c1, c3;
    if ( z > 0 )
      c1 = sharedCost[z - 1];
    else 
      c1 = DSI_HIGHDSI_VALUE;
    
    if ( z < dim.z - 1 ) 
      c3 = sharedCost[z + 1];
    else
      c3 = DSI_HIGHDSI_VALUE;
      
    float m;      
    int p;  
    if ( c1 < c2 && c1 < c3 ) {
      m = c1;
      p = 1;
    } else if ( c2 < c3 ) {
      m = c2;
      p = 0;
    } else { //if ( c3 < c2 && c3 < c1 ){
      m = c3;
      p = -1;
    } 
            
    sumCostDSI[c0Offset] = c0 + m;
    pathDSI[c0Offset] = p;
    
    __syncthreads();
  }
}
#else
__global__ void dynamicprog(int x, const DSIDim dim, const float *costDSI,
                            float *sumCostDSI, int *pathDSI)
{
  //uint z = threadIdx.x + blockIdx.x*blockDim.x;
  //uint y = threadIdx.y + blockIdx.y*blockDim.y;
  const uint z = threadIdx.x;
  const uint y = blockIdx.x;

  if ( z >= dim.z || y >= dim.y  )
    return ;
  
  if ( x == 0 )
    {
  const uint initialOff = dsiOffset(dim, x, y, z);
  sumCostDSI[initialOff] = costDSI[initialOff];
  pathDSI[initialOff] = 0;
    }
  else
    {    
    const uint c0Offset = dsiOffset(dim, x, y, z);  
    const float c0 = costDSI[c0Offset];
      
    /**
     * c1\
     * c2-c0
     * c3/
     */  
    const float c1 = dsiIntensityClamped(dim, x - 1, y, z + 1, sumCostDSI);
    const float c2 = dsiIntensityClamped(dim, x - 1, y, z, sumCostDSI);
    const float c3 = dsiIntensityClamped(dim, x - 1, y, z - 1, sumCostDSI);
    
    float m;      
    int p;  
    if ( c1 < c2 && c1 < c3 ) {
      m = c1;
      p = 1;
    } else if ( c2 < c3 ) {
      m = c2;
      p = 0;
    } else { 
      m = c3;
      p = -1;
    } 
            
    sumCostDSI[c0Offset] = c0 + m;
    pathDSI[c0Offset] = p;
    }

}
#endif

__global__ void reduceImage(const DSIDim dim, const float *sumCostDSI, 
                            const int *pathDSI, float *dispImg)
{
  //const uint y = threadIdx.x + blockIdx.x*blockDim.x; // slice
  const uint y = blockIdx.x;
  if ( y >= dim.y )
    return ;
  
  int lastMinZ = 0;
  float min = dsiIntensityClamped(dim, dim.x - 1, y, 0, sumCostDSI);
  
  for (uint z=1; z < dim.z; z++) {
    const float sc = dsiIntensityClamped(dim, dim.x - 1, y, z, sumCostDSI);    
    if ( sc < min ) {
      lastMinZ = z;
      min = sc;
    }          
  }
  
  uint imgOffset = y*dim.x + (dim.x - 1);
  dispImg[imgOffset] = float(lastMinZ)/float(dim.z);
  
  for (uint _x=0; _x < dim.x - 1; _x++) {
    const uint x = dim.x - 2 - _x;
    const uint offset = dsiOffset(dim, x, y, lastMinZ);
    const int p = pathDSI[offset];
    const uint nz = lastMinZ + p;
    
    if ( nz < dim.maxOffset )
      lastMinZ = nz;
    
    imgOffset = y*dim.x + x;    
    dispImg[imgOffset] = float(lastMinZ)/float(dim.z);      
  }
}

void RunDynamicProgDev(const tdv::Dim &tdv_dsiDim, float *dsi, float *dispImg)
{  
  tdv::CUerrExp cuerr;
  
  DSIDim dsiDim(DSIDimCreate(tdv_dsiDim));  
  tdv::DSIMem sumCostDSI = tdv::DSIMem::Create(tdv_dsiDim);
 
  int *pathDSI;
  cuerr << hipMalloc((void**) &pathDSI, tdv_dsiDim.size()*sizeof(int));
  
  tdv::CudaConstraits constraits;  
  tdv::WorkSize ws = constraits.imageWorkSize(tdv::Dim(tdv_dsiDim.depth(), 
                                                       tdv_dsiDim.width()));  
  tdv::CudaBenchmarker bm;
  bm.begin();
#if SINGLE_KERNEL
  dynamicprog<<<tdv_dsiDim.height(), tdv_dsiDim.depth()>>>(dsiDim, dsi, sumCostDSI.mem(), pathDSI);  
#else
  for (size_t x=0; x<tdv_dsiDim.width(); x++) {
    dynamicprog<<<tdv_dsiDim.height(), tdv_dsiDim.depth()>>>(x, dsiDim, dsi, sumCostDSI.mem(), pathDSI);  
    cuerr << hipDeviceSynchronize();
  }
#endif
  bm.end();
  cuerr << hipDeviceSynchronize();
  
  reduceImage<<<tdv_dsiDim.height(), 1>>>(dsiDim, sumCostDSI.mem(), pathDSI, dispImg);
  
  cuerr << hipFree(pathDSI);  
}