#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include "cuerr.hpp"
#include "dim.hpp"
#include "dsimemutil.h"
#include "cudaconstraits.hpp"

#define MAX_DISP 2048

__global__ void dynamicprog(const dim3 dsiDim, hipPitchedPtr costDSI,
                            hipPitchedPtr pathDSI, float *lastSumCost)
{
  const ushort z = threadIdx.x;
  const ushort y = blockIdx.x;
  const ushort dz = z + 1;
  
  __shared__ float sharedCost[MAX_DISP + 2];
  __shared__ float *costDsiRow;
  __shared__ char *pathDsiRow;
      
  sharedCost[z] = (DSI_GET_ROWF(costDSI, dsiDim, 0, y))[z];
  DSI_GET_ROWB(pathDSI, dsiDim, 0, y)[z] = 0;

  if ( z == 0 ) {    
    sharedCost[0] = HIP_INF_F;
    sharedCost[dsiDim.z + 1] = HIP_INF_F;
  }

  __syncthreads();  
  
  for (ushort x=1; x<dsiDim.x; x++) {                

    if ( z == 0 ) {
      costDsiRow = DSI_GET_ROWF(costDSI, dsiDim, x, y);
      pathDsiRow = DSI_GET_ROWB(pathDSI, dsiDim, x, y);
    }
    
    __syncthreads();
    
    const float c0 = costDsiRow[z];
        
    const float c1 = sharedCost[dz - 1];    
    const float c2 = sharedCost[dz];
    const float c3 = sharedCost[dz + 1];
      
    float m;      
    char p;  
    
    if ( c1 < c2 && c1 < c3 ) {
      m = c1;
      p = -1;
    } else if ( c2 < c3 ) {
      m = c2;
      p = 0;
    } else {
      m = c3;
      p = 1;
    } 
    
    pathDsiRow[z] = p;
    sharedCost[dz] = c0 + m;

    __syncthreads();
  }
  
  lastSumCost[dsiDim.z*y + z] = sharedCost[dz];
}

__global__ void reduceImage(const dim3 dsiDim, 
                            const hipPitchedPtr pathDSI, 
                            const float *lastSumCost,
                            float *dispImg)
{
  const uint y = blockIdx.x;
    
  int lastMinZ = 0;
  float min = lastSumCost[0];
  
  const uint lscBaseOff = y*dsiDim.z;
  for (uint z=1; z < dsiDim.z; z++) {
    const float sc = lastSumCost[lscBaseOff*y + z];
    
    if ( sc < min ) {
      lastMinZ = z;
      min = sc;
    }          
  }
  
  float *imgRow = &dispImg[y*dsiDim.x];  
  imgRow[dsiDim.x - 1] = float(lastMinZ)/float(dsiDim.z);  
  
  for (int x = dsiDim.x - 1; x >= 0; x--) {            
    const char p = dsiGetValueB(pathDSI, dsiDim.y, x, y, lastMinZ);        
    const int nz = lastMinZ + p;
    
    if ( nz >= 0 && nz < dsiDim.z ) {
      lastMinZ = nz;  
    }        
    
    imgRow[x] = float(lastMinZ)/float(dsiDim.z);
  }
  
}

TDV_NAMESPACE_BEGIN

void DynamicProgDevRun(const tdv::Dim &dsiDim, 
                       const hipPitchedPtr costDSI,
                       hipPitchedPtr pathDSI,
                       float *lastSumCosts,
                       float *dispImg)
{  
  CUerrExp cuerr;  
                      
  dynamicprog<<<dsiDim.height(), 
    dsiDim.depth()>>>(tdvDimTo(dsiDim), costDSI, pathDSI, lastSumCosts);
  
  cuerr = hipDeviceSynchronize();
    
  reduceImage<<<dsiDim.height(), 1>>>(tdvDimTo(dsiDim), 
                                      pathDSI, lastSumCosts, 
                                      dispImg);      
}

TDV_NAMESPACE_END