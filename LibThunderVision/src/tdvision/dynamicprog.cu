#include "hip/hip_runtime.h"
#include "dim.hpp"
#include "dsimem.hpp"
#include "dsimemutil.h"
#include "benchmark.hpp"
#include "cudaconstraits.hpp"

#if 0
__global__ void dynamicprog(const DSIDim dim, const float *costDSI,
                            float *sumCostDSI, int *pathDSI)
{
  //uint z = threadIdx.x + blockIdx.x*blockDim.x;
  //uint y = threadIdx.y + blockIdx.y*blockDim.y;
  const uint z = threadIdx.x;
  const uint y = blockIdx.x;

  if ( z >= dim.z || y >= dim.y  )
    return ;
  
  const uint initialOff = dsiOffset(dim, 0, y, z);
  sumCostDSI[initialOff] = costDSI[initialOff];
  pathDSI[initialOff] = 0;

  __syncthreads();

  for (uint x=1; x<dim.x; x++) {    
    __threadfence();
    __syncthreads();
    
    const uint c0Offset = dsiOffset(dim, x, y, z);  
    const float c0 = costDSI[c0Offset];
      
    /**
     * c1\
     * c2-c0
     * c3/
     */  
    const float c1 = dsiIntensityClamped(dim, x - 1, y, z + 1, sumCostDSI);
    const float c2 = dsiIntensityClamped(dim, x - 1, y, z, sumCostDSI);
    const float c3 = dsiIntensityClamped(dim, x - 1, y, z - 1, sumCostDSI);
    
    float m;      
    int p;  
    if ( c1 < c2 && c1 < c3 ) {
      m = c1;
      p = 1;
    } else if ( c2 < c3 ) {
      m = c2;
      p = 0;
    } else { //if ( c3 < c2 && c3 < c1 ){
      m = c3;
      p = -1;
    } 
      
      
    sumCostDSI[c0Offset] = c0 + m;
    pathDSI[c0Offset] = p;
    
    __threadfence();
    __syncthreads();
  }
}
#else
__global__ void dynamicprog(int x, const DSIDim dim, const float *costDSI,
                            float *sumCostDSI, int *pathDSI)
{
  //uint z = threadIdx.x + blockIdx.x*blockDim.x;
  //uint y = threadIdx.y + blockIdx.y*blockDim.y;
  const uint z = threadIdx.x;
  const uint y = blockIdx.x;

  if ( z >= dim.z || y >= dim.y  )
    return ;
  
  if ( x == 0 )
    {
  const uint initialOff = dsiOffset(dim, x, y, z);
  sumCostDSI[initialOff] = costDSI[initialOff];
  pathDSI[initialOff] = 0;
    }
  else
    {    
    const uint c0Offset = dsiOffset(dim, x, y, z);  
    const float c0 = costDSI[c0Offset];
      
    /**
     * c1\
     * c2-c0
     * c3/
     */  
    const float c1 = dsiIntensityClamped(dim, x - 1, y, z + 1, sumCostDSI)*0.90;
    const float c2 = dsiIntensityClamped(dim, x - 1, y, z, sumCostDSI)*0.85;
    const float c3 = dsiIntensityClamped(dim, x - 1, y, z - 1, sumCostDSI);
    
    float m;      
    int p;  
    if ( c1 < c2 && c1 < c3 ) {
      m = c1;
      p = 1;
    } else if ( c2 < c3 ) {
      m = c2;
      p = 0;
    } else { 
      m = c3;
      p = -1;
    } 
            
    sumCostDSI[c0Offset] = c0 + m;
    pathDSI[c0Offset] = p;
    }

}
#endif

__global__ void reduceImage(const DSIDim dim, const float *sumCostDSI, 
                            const int *pathDSI, float *dispImg)
{
  //const uint y = threadIdx.x + blockIdx.x*blockDim.x; // slice
  const uint y = blockIdx.x;
  if ( y >= dim.y )
    return ;
  
  int lastMinZ = 0;
  float min = dsiIntensityClamped(dim, dim.x - 1, y, 0, sumCostDSI);
  
  for (uint z=1; z < dim.z; z++) {
    const float sc = dsiIntensityClamped(dim, dim.x - 1, y, z, sumCostDSI);    
    if ( sc < min ) {
      lastMinZ = z;
      min = sc;
    }          
  }
  
  uint imgOffset = y*dim.x + (dim.x - 1);
  dispImg[imgOffset] = float(lastMinZ)/float(dim.z);
      
  for (uint _x=0; _x < dim.x - 1; _x++) {
    const uint x = dim.x - 2 - _x;
    const uint offset = dsiOffset(dim, x, y, lastMinZ);
    const uint nz = lastMinZ + pathDSI[offset];
    
    if ( nz < dim.maxOffset )
      lastMinZ = nz;
    
    imgOffset = y*dim.x + x;
    dispImg[imgOffset] = float(lastMinZ)/float(dim.z);    
  }
}

void RunDynamicProgDev(const tdv::Dim &tdv_dsiDim, float *dsi, float *dispImg)
{  
  tdv::CUerrExp cuerr;
  
  DSIDim dsiDim(DSIDimCreate(tdv_dsiDim));  
  tdv::DSIMem sumCostDSI = tdv::DSIMem::Create(tdv_dsiDim);
 
  int *pathDSI;
  cuerr << hipMalloc((void**) &pathDSI, tdv_dsiDim.size()*sizeof(int));
  
  tdv::CudaConstraits constraits;  
  tdv::WorkSize ws = constraits.imageWorkSize(tdv::Dim(tdv_dsiDim.depth(), 
                                                       tdv_dsiDim.width()));  
  tdv::CudaBenchmarker bm;
  bm.begin();
#if 0
  dynamicprog<<<tdv_dsiDim.height(), tdv_dsiDim.depth()>>>(dsiDim, dsi, sumCostDSI.mem(), pathDSI);  
#else
  for (int x=0; x<tdv_dsiDim.width(); x++) {
    dynamicprog<<<tdv_dsiDim.height(), tdv_dsiDim.depth()>>>(x, dsiDim, dsi, sumCostDSI.mem(), pathDSI);  
    cuerr << hipDeviceSynchronize();
  }
#endif
  bm.end();
  cuerr << hipDeviceSynchronize();
  
  reduceImage<<<tdv_dsiDim.height(), 1>>>(dsiDim, sumCostDSI.mem(), pathDSI, dispImg);
  
  cuerr << hipFree(pathDSI);  
}