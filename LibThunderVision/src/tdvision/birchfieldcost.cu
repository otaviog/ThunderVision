#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include "cuerr.hpp"
#include "cudaconstraits.hpp"
#include "dsimemutil.h"

texture<float, 2> texLeftImg;
texture<float, 2> texRightImg;

#define min3(a, b, c) min(a, min(b, c))
#define max3(a, b, c) max(a, max(b, c))

__device__ float costAtDisp(int x, int y, int disp)
{
  float sum = 0.0f;

  for (int v=x; v < x + 8; v++) {
    const float lI = tex2D(texLeftImg, v, y);
    const float rI = tex2D(texRightImg, v - disp, y);

    const float laI = 0.5f*(lI + tex2D(texLeftImg, v - 1, y));
    const float lbI = 0.5f*(lI + tex2D(texLeftImg, v + 1, y));

    const float raI = 0.5f*(rI + tex2D(texRightImg, v - disp - 1, y));
    const float rbI = 0.5f*(rI + tex2D(texRightImg, v - disp + 1, y));

    const float lImi = min3(laI, lbI, lI);
    const float lIma = max3(laI, lbI, lI);

    const float rImi = min3(raI, rbI, rI);
    const float rIma = max3(raI, rbI, rI);

    sum += min(max3(0.0f, lI - rIma, rImi - lI),
               max3(0.0f, rI - lIma, lImi - rI));
  }

  return sum;
}

__global__ void birchfieldKernTexture(const dim3 dsiDim, hipPitchedPtr costDSI)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if ( x < dsiDim.x && y < dsiDim.y ) {
    float *costDsiRow = dsiGetRow(costDSI, dsiDim.y, x, y);

    for (int disp=0; disp < dsiDim.z; disp++) {
      float cost = HIP_INF_F;

      if ( x - disp >= 0 ) {
        cost = costAtDisp(x, y, disp);
      }

      costDsiRow[disp] = cost;
    }
  }
}

#define min3(a, b, c) min(a, min(b, c))
#define max3(a, b, c) max(a, max(b, c))

#define MAX_LINE_SIZE 1024
#define BT_LEFT_N 4
#define BT_RIGHT_N 4

__global__ void birchfieldKernSharedMem(const dim3 dsiDim,
                                        const float *leftImg, const float *rightImg,
                                        hipPitchedPtr costDSI)
{
  const short x = threadIdx.x;
  const ushort y = blockIdx.x;

  __shared__ float leftScanLine[MAX_LINE_SIZE + 2];
  __shared__ float rightScanLine[MAX_LINE_SIZE + 2];

  const uint cPixelOffset = y*dsiDim.x + x;

  const short nx = x + 1;
    
  leftScanLine[nx] = leftImg[cPixelOffset];
  rightScanLine[nx] = rightImg[cPixelOffset];

  if ( x == 0 ) {
    leftScanLine[0] = 0.0f;
    rightScanLine[0] = 0.0f;

    leftScanLine[dsiDim.x + 1] = 0.0f;
    rightScanLine[dsiDim.x + 1] = 0.0f;
  }

  __syncthreads();

  float *costDsiRow = dsiGetRow(costDSI, dsiDim.y, x, y);

  const short nDisps = min(dsiDim.z, x + 1);
  for (short disp=0; disp < nDisps; disp++) {
    const short start = max(0, x - disp - BT_LEFT_N) + disp;
    const short end = min(dsiDim.x, x + BT_RIGHT_N);

    float costValue = 0.0f;

    for (ushort p=start; p < end; p++) {
      const ushort lIdx = p + 1;
      const ushort rIdx = p - disp + 1;

      const float lI = leftScanLine[lIdx];
      const float rI = rightScanLine[rIdx];

#if 1
      const float laI = 0.5f*(lI + leftScanLine[lIdx - 1]);
      const float lbI = 0.5f*(lI + leftScanLine[lIdx + 1]);

      const float raI = 0.5f*(rI + rightScanLine[rIdx - 1]);
      const float rbI = 0.5f*(rI + rightScanLine[rIdx + 1]);
#else
      const float laI = 0.5f*(lI + leftScanLine[lIdx]);
      const float lbI = 0.5f*(lI + leftScanLine[lIdx]);

      const float raI = 0.5f*(rI + rightScanLine[rIdx]);
      const float rbI = 0.5f*(rI + rightScanLine[rIdx]);

#endif
      const float lImi = min3(laI, lbI, lI);
      const float lIma = max3(laI, lbI, lI);

      const float rImi = min3(raI, rbI, rI);
      const float rIma = max3(raI, rbI, rI);

      costValue += min(max3(0.0f, lI - rIma, rImi - lI),
                       max3(0.0f, rI - lIma, lImi - rI));
    }

    costDsiRow[disp] = costValue;
  }

  for (short disp=nDisps; disp<dsiDim.z; disp++) {
    costDsiRow[disp] = HIP_INF_F;
  }
}

__global__ void birchfieldKernSharedMem2(const dim3 dsiDim,
                                        hipPitchedPtr costDSI)
{
  const int x = threadIdx.x;
  const int y = blockIdx.x;

  __shared__ float leftScanLine[MAX_LINE_SIZE + 2];
  __shared__ float leftInterp[MAX_LINE_SIZE*2];

  __shared__ float rightScanLine[MAX_LINE_SIZE + 2];
  __shared__ float rightInterp[MAX_LINE_SIZE*2];
  
  const int nx = x + 1;
  const int dimz = dsiDim.z;

  leftScanLine[nx] = tex2D(texLeftImg, x, y);
  rightScanLine[nx] = tex2D(texRightImg, x, y);

  if ( x == 0 ) {
    leftScanLine[0] = 0.0f;
    rightScanLine[0] = 0.0f;

    leftScanLine[dsiDim.x + 1] = 0.0f;
    rightScanLine[dsiDim.x + 1] = 0.0f;
  }

  __syncthreads();

  {
    const int off = x*2;
    const float lI = leftScanLine[nx];
    const float rI = rightScanLine[nx];

    leftInterp[off] = 0.5f*(lI + leftScanLine[nx - 1]);
    rightInterp[off] = 0.5f*(rI + rightScanLine[nx - 1]);

    leftInterp[off + 1] = 0.5f*(lI + leftScanLine[nx + 1]);
    rightInterp[off + 1] = 0.5f*(rI + rightScanLine[nx + 1]);
  }

  __syncthreads();

  float *costDsiRow = dsiGetRow(costDSI, dsiDim.y, x, y);

  const int nDisps = min(dimz, x + 1);
  for (int disp=0; disp < nDisps; disp++) {
    const int start = max(0, x - disp - BT_LEFT_N) + disp;
    const int end = min(dsiDim.x, x + BT_RIGHT_N);

    float costValue = 0.0f;

    for (int p=start; p < end; p++) {
      const int lIdx = p + 1;
      const int rIdx = p - disp + 1;

      const float lI = leftScanLine[lIdx];
      const float rI = rightScanLine[rIdx];

      const float laI = leftInterp[lIdx*2];
      const float lbI = leftInterp[lIdx*2 + 1];

      const float raI = rightInterp[lIdx*2];
      const float rbI = rightInterp[lIdx*2 + 1];

      const float lImi = min3(laI, lbI, lI);
      const float lIma = max3(laI, lbI, lI);

      const float rImi = min3(raI, rbI, rI);
      const float rIma = max3(raI, rbI, rI);

      costValue += min(max3(0.0f, lI - rIma, rImi - lI),
                       max3(0.0f, rI - lIma, lImi - rI));
    }

    costDsiRow[disp] = costValue;
  }

  for (int disp=nDisps; disp<dimz; disp++) {
    costDsiRow[disp] = HIP_INF_F;
  }
}

TDV_NAMESPACE_BEGIN

static void TextureBirchfieldRun(Dim dsiDim,
                                 float *leftImg_d, float *rightImg_d,
                                 hipPitchedPtr dsiMem)
{
  CUerrExp err;
  err << hipBindTexture2D(NULL, texLeftImg, leftImg_d,
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));

  err << hipBindTexture2D(NULL, texRightImg, rightImg_d,
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));

  texLeftImg.addressMode[0] = texRightImg.addressMode[0] = hipAddressModeWrap;
  texLeftImg.addressMode[1] = texRightImg.addressMode[1] = hipAddressModeWrap;
  texLeftImg.normalized = texRightImg.normalized = false;
  texLeftImg.filterMode = texRightImg.filterMode = hipFilterModePoint;

  CudaConstraits constraits;
  WorkSize ws = constraits.imageWorkSize(dsiDim);
  birchfieldKernTexture<<<ws.blocks, ws.threads>>>(tdvDimTo(dsiDim), dsiMem);
}

static void SharedMemBirchfieldRun(Dim dsiDim,
                                   float *leftImg_d, float *rightImg_d,
                                   hipPitchedPtr dsiMem)
{
  CUerrExp err;  
  err << hipBindTexture2D(NULL, texLeftImg, leftImg_d,
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));

  err << hipBindTexture2D(NULL, texRightImg, rightImg_d,
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));

  texLeftImg.addressMode[0] = texRightImg.addressMode[0] = hipAddressModeWrap;
  texLeftImg.addressMode[1] = texRightImg.addressMode[1] = hipAddressModeWrap;
  texLeftImg.normalized = texRightImg.normalized = false;
  texLeftImg.filterMode = texRightImg.filterMode = hipFilterModePoint;

  birchfieldKernSharedMem<<<dsiDim.height(),
    dsiDim.width()>>>(tdvDimTo(dsiDim), leftImg_d, rightImg_d, dsiMem);
}

void BirchfieldCostRun(Dim dsiDim,
                       float *leftImg_d, float *rightImg_d,
                       hipPitchedPtr costDSI)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    if ( static_cast<int>(dsiDim.width()) <= prop.maxThreadsPerBlock ) {
      //    if ( false ) {
      SharedMemBirchfieldRun(dsiDim, leftImg_d, rightImg_d,
                             costDSI);
    }
    else {
      TextureBirchfieldRun(dsiDim, leftImg_d, rightImg_d,
                           costDSI);
    }
    CUerrExp err;
    err << hipDeviceSynchronize();
}

TDV_NAMESPACE_END
