#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include "cuerr.hpp"
#include "cudaconstraits.hpp"
#include "dsimemutil.h"

texture<float, 2> texLeftImg;
texture<float, 2> texRightImg;

#define min3(a, b, c) min(a, min(b, c))
#define max3(a, b, c) max(a, max(b, c))

__device__ float costAtDisp(int x, int y, int disp)
{    
  float sum = 0.0f;
  
  for (int v=x; v < x + 8; v++) {    
    const float lI = tex2D(texLeftImg, v, y);      
    const float rI = tex2D(texRightImg, v - disp, y);   
      
    const float laI = 0.5f*(lI + tex2D(texLeftImg, v - 1, y));
    const float lbI = 0.5f*(lI + tex2D(texLeftImg, v + 1, y));
  
    const float raI = 0.5f*(rI + tex2D(texRightImg, v - disp - 1, y));
    const float rbI = 0.5f*(rI + tex2D(texRightImg, v - disp + 1, y));

    const float lImi = min3(laI, lbI, lI);
    const float lIma = max3(laI, lbI, lI);

    const float rImi = min3(raI, rbI, rI);
    const float rIma = max3(raI, rbI, rI);
    
    sum += min(max3(0.0f, lI - rIma, rImi - lI),
               max3(0.0f, rI - lIma, lImi - rI));
  }
  
  return sum;
}

__global__ void birchfieldKernTexture(const dim3 dsiDim, hipPitchedPtr costDSI)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;     

  if ( x < dsiDim.x && y < dsiDim.y ) {
    float *costDsiRow = dsiGetRow(costDSI, dsiDim.x, x, y);
    
    for (int disp=0; disp < dsiDim.z; disp++) {   
      float cost = HIP_INF_F;
      
      if ( x - disp >= 0 ) {       
        cost = costAtDisp(x, y, disp); 
      }
      
      costDsiRow[disp] = cost;                
    }        
  }
}

#define min3(a, b, c) min(a, min(b, c))
#define max3(a, b, c) max(a, max(b, c))

#define MAX_LINE_SIZE 1024
#define BT_N 4

__global__ void birchfieldKernSharedMem(const dim3 dsiDim, 
                                        const float *leftImg, const float *rightImg,
                                        hipPitchedPtr costDSI)
{
  const uint x = threadIdx.x;
  const uint y = blockIdx.x;
  
  __shared__ float leftScanLine[MAX_LINE_SIZE + 2];
  __shared__ float rightScanLine[MAX_LINE_SIZE + 2];
  
  const uint cPixelOffset = y*dsiDim.x + x;
  const uint dx = x + 1;
  
  leftScanLine[dx] = leftImg[cPixelOffset];
  rightScanLine[dx] = rightImg[cPixelOffset];
  
  if ( x == 0 ) {
    leftScanLine[0] = 0.0f;
    rightScanLine[0] = 0.0f;    
  }
  
  __syncthreads();      
  
  float *costDsiRow = dsiGetRow(costDSI, dsiDim.x, x, y);
  
  for (int disp=0; disp < dsiDim.z; disp++) {   
    float costValue = HIP_INF_F;    
    
    if ( static_cast<int>(x) - disp >= 0 ) {       
      costValue = 0.0f;      
      
      for (uint v=dx; v < dx + BT_N; v++) {  
        const uint vd = v - disp;
          
        const float lI = leftScanLine[dx];
        const float rI = rightScanLine[vd];  
      
        const float laI = 0.5f*(lI + leftScanLine[dx - 1]);
        const float lbI = 0.5f*(lI + leftScanLine[dx + 1]);
  
        const float raI = 0.5f*(rI + rightScanLine[vd - 1]);
        const float rbI = 0.5f*(rI + rightScanLine[vd + 1]);

        const float lImi = min3(laI, lbI, lI);
        const float lIma = max3(laI, lbI, lI);

        const float rImi = min3(raI, rbI, rI);
        const float rIma = max3(raI, rbI, rI);
    
        costValue += min(max3(0.0f, lI - rIma, rImi - lI),
                         max3(0.0f, rI - lIma, lImi - rI));              
      }            
    }
    
    costDsiRow[disp] = costValue;
  }          
}

TDV_NAMESPACE_BEGIN

static void TextureBirchfieldRun(Dim dsiDim, 
                                 float *leftImg_d, float *rightImg_d,
                                 hipPitchedPtr dsiMem)
{
  CUerrExp err;  
  err << hipBindTexture2D(NULL, texLeftImg, leftImg_d, 
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));
  
  err << hipBindTexture2D(NULL, texRightImg, rightImg_d, 
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));
  
  texLeftImg.addressMode[0] = texRightImg.addressMode[0] = hipAddressModeWrap;
  texLeftImg.addressMode[1] = texRightImg.addressMode[1] = hipAddressModeWrap;
  texLeftImg.normalized = texRightImg.normalized = false;
  texLeftImg.filterMode = texRightImg.filterMode = hipFilterModePoint;
        
  CudaConstraits constraits;  
  WorkSize ws = constraits.imageWorkSize(dsiDim);  
  birchfieldKernTexture<<<ws.blocks, ws.threads>>>(tdvDimTo(dsiDim), dsiMem);
}

static void SharedMemBirchfieldRun(Dim dsiDim,
                                   float *leftImg_d, float *rightImg_d,
                                   hipPitchedPtr dsiMem)
{
  CUerrExp err;  
  birchfieldKernSharedMem<<<dsiDim.height(), 
    dsiDim.width()>>>(tdvDimTo(dsiDim), leftImg_d, rightImg_d, dsiMem); 
}

void BirchfieldCostRun(Dim dsiDim,
                       float *leftImg_d, float *rightImg_d,
                       hipPitchedPtr costDSI)
{
    hipDeviceProp_t prop;    
    hipGetDeviceProperties(&prop, 0);
    
    if ( dsiDim.width() <= prop.maxThreadsPerBlock ) {
      SharedMemBirchfieldRun(dsiDim, leftImg_d, rightImg_d,
                             costDSI);      
    }
    else {
      TextureBirchfieldRun(dsiDim, leftImg_d, rightImg_d,
                           costDSI);
    }

    hipDeviceSynchronize();
}

TDV_NAMESPACE_END
