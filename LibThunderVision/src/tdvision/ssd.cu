#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuerr.hpp"
#include "cudaconstraits.hpp"
#include "dsimemutil.h"

texture<float, 2> texLeftImg;
texture<float, 2> texRightImg;

#define SSD_WIND_DIM 5
#define SSD_WIND_START -3
#define SSD_WIND_END 4

__device__ float ssdAtDisp(int x, int y, int disp)
{
  float sum = 0.0f;
  
  for (int row=SSD_WIND_START; row<SSD_WIND_END; row++)
    for (int col=SSD_WIND_START; col<SSD_WIND_END; col++) {
      
      float lI = tex2D(texLeftImg, x + col, y + row), 
        rI = tex2D(texRightImg, x + col - disp, y + row);   
      
      sum += (lI - rI)*(lI - rI);      
    }
  
  return sum;
}

__global__ void ssdKern(const DSIDim dsiDim, const int maxDisparity, 
                        float *dsiMem)
{  
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;     

  if ( x < dsiDim.x && y < dsiDim.y ) {
    
    for (int disp=0; (disp < maxDisparity); disp++) {   
      float ssdValue = HIP_INF_F; 
      if ( x - disp >= 0 ) {       
        ssdValue = ssdAtDisp(x, y, disp);                    
      }
      
      dsiSetIntensity(dsiDim, x, y, disp, ssdValue, dsiMem);
    }    
    
  }
}

TDV_NAMESPACE_BEGIN

void DevSSDRun(Dim dsiDim, int maxDisparity, 
               float *leftImg_d, float *rightImg_d,
               float *dsiMem)
{
  CUerrExp err;
  
  err << hipBindTexture2D(NULL, texLeftImg, leftImg_d, 
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));
  
  err << hipBindTexture2D(NULL, texRightImg, rightImg_d, 
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));
  
  texLeftImg.addressMode[0] = texRightImg.addressMode[0] = hipAddressModeWrap;
  texLeftImg.addressMode[1] = texRightImg.addressMode[1] = hipAddressModeWrap;
  texLeftImg.normalized = texRightImg.normalized = false;
  texLeftImg.filterMode = texRightImg.filterMode = hipFilterModePoint;
    
  DSIDim ddim(DSIDimCreate(dsiDim));  
  CudaConstraits constraits;  
  WorkSize ws = constraits.imageWorkSize(dsiDim);  
  ssdKern<<<ws.blocks, ws.threads>>>(ddim, maxDisparity, dsiMem);     
}

TDV_NAMESPACE_END