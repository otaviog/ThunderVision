#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include "cuerr.hpp"
#include "cudaconstraits.hpp"
#include "dsimemutil.h"

texture<float, 2> texLeftImg;
texture<float, 2> texRightImg;

#define SSD_WIND_DIM 5
#define SSD_WIND_START -3
#define SSD_WIND_END 4

inline __host__ __device__ uint dsiOffset(const dim3 dim, uint x, uint y, uint z)
{
    return dim.z*dim.y*x + dim.z*y + z;
}

inline __host__ __device__ void dsiSetIntensity(const dim3 dim, uint x, uint y, uint z, float value, 
    float *dsi)
{
    dsi[dsiOffset(dim, x, y, z)] = value;
}

__device__ float ssdAtDisp(int x, int y, int disp)
{
  float sum = 0.0f;

  for (int row=SSD_WIND_START; row<SSD_WIND_END; row++)
    for (int col=SSD_WIND_START; col<SSD_WIND_END; col++) {

      float lI = tex2D(texLeftImg, x + col, y + row),
        rI = tex2D(texRightImg, x + col - disp, y + row);

      sum += (lI - rI)*(lI - rI);
    }

  return sum;
}

__global__ void ssdKern(const dim3 dsiDim, hipPitchedPtr dsiMem)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y; 
  
  if ( x < dsiDim.x && y < dsiDim.y ) {
    float *dsiRow = (float*) (((char *) dsiMem.ptr) + dsiMem.pitch*dsiDim.x*y
                              + dsiMem.pitch*x);

    for (int disp=0; disp < dsiDim.z; disp++) {
      float ssdValue = HIP_INF_F;

      if ( x - disp >= 0 ) {
        ssdValue = ssdAtDisp(x, y, disp);
      }
      
      dsiSetIntensity(dsiDim, x, y, disp, ssdValue, (float*) dsiMem.ptr);
      //dsiRow[disp] = ssdValue;
    }
  }
}

TDV_NAMESPACE_BEGIN

void SSDDevRun(Dim dsiDim, float *leftImg_d, float *rightImg_d,
               hipPitchedPtr dsiMem)
{
  CUerrExp err;

  err << hipBindTexture2D(NULL, texLeftImg, leftImg_d,
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));

  err << hipBindTexture2D(NULL, texRightImg, rightImg_d,
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));

  texLeftImg.addressMode[0] = texRightImg.addressMode[0] = hipAddressModeWrap;
  texLeftImg.addressMode[1] = texRightImg.addressMode[1] = hipAddressModeWrap;
  texLeftImg.normalized = texRightImg.normalized = false;
  texLeftImg.filterMode = texRightImg.filterMode = hipFilterModePoint;

  CudaConstraits constraits;
  WorkSize ws = constraits.imageWorkSize(dsiDim);
  ssdKern<<<ws.blocks, ws.threads>>>(tdvDimTo(dsiDim), dsiMem);

  err << hipDeviceSynchronize();
}

TDV_NAMESPACE_END