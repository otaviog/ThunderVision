#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuerr.hpp"
#include "cudaconstraits.hpp"

texture<float, 2> texLeftImg;
texture<float, 2> texRightImg;

__device__ float ssdAtDisp(int x, int y, int disp)
{
  float sum = 0.0f;
  
  for (int row=-1; row<2; row++)
    for (int col=-1; col<2; col++) {
      
      float lI = tex2D(texLeftImg, x + col, y + row), 
        rI = tex2D(texRightImg, x + disp + col, y + row);   
      
      sum += (lI - rI)*(lI - rI);
    }
  
  return sum;
}

__global__ void ssdKern(const int maxDisparity, const dim3 dim, float *dsiMem)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;     

  if ( x < dim.x && y < dim.y ) {    
    for (int disp=0; (disp < maxDisparity) && (x + disp) < dim.x; disp++) {   
      float ssdValue = ssdAtDisp(x, y, disp);

      const int volOffset = (dim.x*dim.y)*disp + y*dim.x + x;
      // volOffset =  disp + dim.z*x + (dim.z*dim.x)*y;
      dsiMem[volOffset] = ssdValue;
    }    
    
  }
}

TDV_NAMESPACE_BEGIN

void DevSSDRun(int maxDisparity,
               Dim dsiDim, float *leftImg_d, float *rightImg_d,
               float *dsiMem)
{
  CUerrExp err;
    
  err << hipBindTexture2D(NULL, texLeftImg, leftImg_d, 
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));
  
  err << hipBindTexture2D(NULL, texRightImg, rightImg_d, 
                           hipCreateChannelDesc<float>(),
                           dsiDim.width(), dsiDim.height(),
                           dsiDim.width()*sizeof(float));
  
  texLeftImg.addressMode[0] = texRightImg.addressMode[0] = hipAddressModeWrap;
  texLeftImg.addressMode[1] = texRightImg.addressMode[1] = hipAddressModeWrap;
  texLeftImg.normalized = texRightImg.normalized = false;
  texLeftImg.filterMode = texRightImg.filterMode = hipFilterModePoint;
    
  CudaConstraits constraits;  
  WorkSize ws = constraits.imageWorkSize(dsiDim);
  
  ssdKern<<<ws.blocks, ws.threads>>>(maxDisparity, 
                                     dim3(dsiDim.width(), dsiDim.height(),
                                          dsiDim.depth()),
                                     dsiMem); 
}

TDV_NAMESPACE_END